
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <limits.h>

/* GPU */
__global__ void find_odd(int n, int *A, int *B) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < n; i += stride) {
        if (A[i] % 2 > 0) { B[i] = A[i]; }
        else { B[i] = 0; }
    }
}

int main() {
    
    /* Open File */
    printf("Opening File!\n");
    FILE *fp;
    fp = fopen("inp.txt", "r");
    
    printf("Init Arrays and Such!\n");
    char buff[256];
    const int M = 1<<20;
    int *A = new int[M];
    int *B = new int[M];
    int *D = new int[M];
    int i, count = 0;
    
    /* Copy to GPU Memory */
    printf("Copying to GPU Memory!\n");
    hipMallocManaged(&A, M * sizeof(int));
    hipMallocManaged(&B, M * sizeof(int));
    
    /* Read numbers as integers one by one */
    printf("Scanning File!\n");
    while (fscanf(fp, "%d", &i) != EOF) {
        A[count++] = i;             // Add number to array
        fscanf(fp, "%s", buff);     // Read until whitespace
    }
    
    /* Close File */
    printf("Closing File!\n");
    fclose(fp);
    
    /* Kernel */
    printf("Accessing GPU!\n");
    int blockSize = 256;
    int numBlocks = (count + blockSize - 1) / blockSize;
    find_odd<<<numBlocks, blockSize>>>(count, A, B);
    
    /* Wait for GPU */
    hipDeviceSynchronize();
    
    /* Print B */
    for (int i = 0; i < count; i++) {
        printf("%d, ", B[i]);
    }
    printf("\n");

    /* Remove 0s */
    printf("Removing Zeros!\n");
    int zeroCount = 0;
    for (int i = 0; i < count; i++) {
        if (B[i] != 0) { D[i - zeroCount] = B[i]; }
        else { zeroCount++; }
    }

    /* Print D */
    printf("Printing Array!\n");
    for (int i = 0; D[i] != 0; i++) {
        printf("%d", D[i]);
        if (D[i + 1] != 0) printf(", ");
    }
    printf("\n");

    /* Write D */
    printf("Writing File!\n");
    FILE *f = fopen("q3.txt", "w");
    for (int i = 0; D[i] != 0; i++) {
        fprintf(f, "%d", D[i]);
        if (D[i + 1] != 0) fprintf(f, ", ");
    }
    fclose(f);

    /* Free Memory */
    printf("Freeing Memory!\n");
    hipFree(A);
    hipFree(B);
    
    return 0;
}
