
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <limits.h>

/* GPU */
__global__ void last_digit(int n, int *A, int *D) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < n; i += stride) {
        if (A[i] % 2 == 1) { D[i] = A[i]; }
        else { D[i] = 0; }
    }
}

/* Remove Values in Array */
int* remove_copy(const int *in, size_t n, int *out, int value) {
    for (size_t i = 0; i != n; i++)
        if (in[i] != value) *out++ = in[i];
    return out;
}

int main() {
    
    /* Open File */
    printf("Opening File!\n");
    FILE *fp;
    fp = fopen("inp.txt", "r");
    
    printf("Init Arrays and Such!\n");
    char buff[256];
    const int M = 1<<20;
    int *A = new int[M];
    int *D = new int[M];
    int i, count = 0;
    
    /* Read numbers as integers one by one */
    printf("Scanning File!\n");
    while (fscanf(fp, "%d", &i) != EOF) {
        A[count++] = i;             // Add number to array
        fscanf(fp, "%s", buff);     // Read until whitespace
    }
    
    /* Close File */
    printf("Closing File!\n");
    fclose(fp);
    
    /* Copy to GPU Memory */
    printf("Copying to GPU Memory!\n");
    hipMallocManaged(&A, M * sizeof(int));
    hipMallocManaged(&D, M * sizeof(int));
    
    /* Kernel */
    printf("Accessing GPU!\n");
    int blockSize = 256;
    int numBlocks = (count + blockSize - 1) / blockSize;
    last_digit<<<numBlocks, blockSize>>>(count, A, D);
    
    /* Remove 0s */
    printf("Removing Zeros!\n");
    int *B = new int[sizeof(D) / sizeof(*D)];
    const size_t N = sizeof(D) / sizeof(*D);
    int *done = remove_copy(D, N, B, 0);
    
    /* Print Array */
    int length = sizeof(done) / sizeof(*done);
    for (int i = 0; i < length; i++) {
        printf("%d", done[i]);
        if (i + 1 != length) { printf(", "); }
    }
    
    /* Write Out */
    FILE *f = fopen("q3.txt", "w");
    for (int i = 0; i < length; i++) {
        fprintf(f, "%d", done[i]);
        if (i + 1 != length) { fprintf(f, ", "); }
    } fclose(f);
    
    /* Free Memory */
    hipFree(A);
    hipFree(D);
    
    return 0;
}
