#include "hip/hip_runtime.h"
#include <stdio.h>
#include <limits.h>

/* GPU */
__global__ void last_digit(int n, int *A, int *D) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < n; i += stride) {
        if (A[i] % 2 == 1) { D[i] = A[i]; }
        else { D[i] = 0; }
    }
}

/* Remove Values in Array */
int* remove_copy(const int *in, size_t n, int *out, int value) {
    for (size_t i = 0; i != n; i++)
        if (in[i] != value) *out++ = in[i];
    return out;
}

int main() {
    
    /* Open File */
    FILE *fp;
    fp = fopen("inp.txt", "r");
    
    char buff[256];
    const int M = 1<<20;
    int *A = new int[M];
    int *D = new int[M];
    int i, count = 0;
    
    /* Copy to GPU Memory */
    hipMallocManaged(&A, M * sizeof(int));
    hipMallocManaged(&D, M * sizeof(int));
    
    /* Read numbers as integers one by one */
    while (fscanf(fp, "%d", &i) != EOF) {
        A[count++] = i;             // Add number to array
        fscanf(fp, "%s", buff);     // Read until whitespace
    }
    
    /* Close FilePointer */
    fclose(fp);
    
    /* Kernel */
    int blockSize = 256;
    int numBlocks = (count + blockSize - 1) / blockSize;
    last_digit<<<numBlocks, blockSize>>>(count, A, D);
    
    /* Remove 0s */
    const size_t N = sizeof(A) / sizeof(*A);
    int *done = remove_copy(A, N, D, 0);
    
    /* Print Array */
    for (int i = 0; i < length; i++) {
        printf(f, "%d", done[i]);
        if (i + 1 != length) { printf(f, ", "); }
    }
    
    /* Write Out */
    int length = sizeof(done) / sizeof(*done);
    FILE *f = fopen("q3.txt", "w");
    for (int i = 0; i < length; i++) {
        fprintf(f, "%d", done[i]);
        if (i + 1 != length) { fprintf(f, ", "); }
    } fclose(f);
    
    /* Free Memory */
    hipFree(A);
    hipFree(D);
    
    return 0;
}
