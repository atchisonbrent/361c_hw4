
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <limits.h>

/* GPU */
__global__ void last_digit(int n, int *A, int *D) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < n; i += stride) {
        if (A[i] % 2 == 1) { printf("Value is odd!\n"); D[i] = A[i]; }
        else { printf("Value is even!\n"); D[i] = 0; }
    }
}

/* Remove Values in Array */
int* remove_copy(const int *in, size_t n, int *out, int value) {
    for (size_t i = 0; i != n; i++)
        if (in[i] != value) *out++ = in[i];
    return out;
}

int main() {
    
    /* Open File */
    printf("Opening File!\n");
    FILE *fp;
    fp = fopen("inp.txt", "r");
    
    printf("Init Arrays!\n");
    char buff[256];
    const int M = 1<<20;
    int *A = new int[M];
    int *D = new int[M];
    int i, count = 0;
    
    /* Copy to GPU Memory */
    printf("Copying to GPU Memory!\n");
    hipMallocManaged(&A, M * sizeof(int));
    hipMallocManaged(&D, M * sizeof(int));
    
    /* Read numbers as integers one by one */
    printf("Scanning File!\n");
    while (fscanf(fp, "%d", &i) != EOF) {
        A[count++] = i;             // Add number to array
        fscanf(fp, "%s", buff);     // Read until whitespace
    }
    
    /* Close File */
    printf("Closing File!\n");
    fclose(fp);
    
    /* Kernel */
    printf("Accessing GPU!\n");
    int blockSize = 256;
    int numBlocks = (count + blockSize - 1) / blockSize;
    last_digit<<<numBlocks, blockSize>>>(count, A, D);
    
    /* Remove 0s */
    printf("Removing Zeros!\n");
    const size_t N = sizeof(A) / sizeof(*A);
    int *done = remove_copy(A, N, D, 0);
    
    /* Print Array */
    int length = sizeof(done) / sizeof(*done);
    for (int i = 0; i < length; i++) {
        printf("%d", done[i]);
        if (i + 1 != length) { printf(", "); }
    }
    
    /* Write Out */
    FILE *f = fopen("q3.txt", "w");
    for (int i = 0; i < length; i++) {
        fprintf(f, "%d", done[i]);
        if (i + 1 != length) { fprintf(f, ", "); }
    } fclose(f);
    
    /* Free Memory */
    hipFree(A);
    hipFree(D);
    
    return 0;
}
