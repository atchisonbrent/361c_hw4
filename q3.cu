
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <limits.h>

/* GPU */
__global__ void find_odd(int n, int *A, int *B) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < n; i += stride) {
        if (A[i] % 2 == 1) { B[i] = A[i]; }
        else { B[i] = 0; }
    }
}

int main() {
    
    /* Open File */
    printf("Opening File!\n");
    FILE *fp;
    fp = fopen("inp.txt", "r");
    
    printf("Init Arrays and Such!\n");
    char buff[256];
    const int M = 1<<20;
    int *A = (int*) malloc(M * sizeof(int));
    int *B = (int*) malloc(M * sizeof(int));
    int *D = (int*) malloc(M * sizeof(int));
//    int *A = new int[M];
//    int *B = new int[M];
//    int *D = new int[M];
    int i, count = 0;
    
    /* Read numbers as integers one by one */
    printf("Scanning File!\n");
    while (fscanf(fp, "%d", &i) != EOF) {
        A[count++] = i;             // Add number to array
        fscanf(fp, "%s", buff);     // Read until whitespace
    }
    
    /* Close File */
    printf("Closing File!\n");
    fclose(fp);
    
    /* Copy to GPU Memory */
    printf("Copying to GPU Memory!\n");
    hipMallocManaged(&A, M * sizeof(int));
    hipMallocManaged(&B, M * sizeof(int));
    
    /* Kernel */
    printf("Accessing GPU!\n");
    int blockSize = 256;
    int numBlocks = (count + blockSize - 1) / blockSize;
    find_odd<<<numBlocks, blockSize>>>(count, A, B);

    /* Remove 0s */
    printf("Removing Zeros!\n");
    int zeroCount = 0;
    for (int i = 0; i < count; i++) {
        if (B[i] == 0) { zeroCount++; }
        else { D[i - zeroCount] = B[i]; }
    }

    /* Print Array */
    printf("Printing Array!\n");
    for (int i = 0; D[i] != 0; i++) { printf("%d, ", D[i]); }

    /* Write Out */
    printf("Writing File!\n");
    FILE *f = fopen("q3.txt", "w");
    for (int i = 0; D[i] != 0; i++) { fprintf(f, "%d, ", D[i]); }
    fclose(f);

    /* Free Memory */
    printf("Freeing Memory!\n");
    hipFree(A);
    hipFree(B);
    
    return 0;
}
