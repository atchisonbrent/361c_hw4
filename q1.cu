#include "hip/hip_runtime.h"
#include <stdio.h>
#include <limits.h>
#include <hip/hip_runtime.h>

__global__ void min(int n, int *A) {
    
    __shared__ int min = INT_MAX;
    
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < n; i += stride)
        atomicMin(min, A[i]);
}

int main() {
    
    /* Open File */
    FILE *fp;
    fp = fopen("inp.txt", "r");
    
    char buff[256];
    const int M = 1 << 20;          // 1 Million
    int *A = new int[M];
    int i, count = 0;
    
    hipMallocManaged(&A, M * sizeof(int));

    /* Read numbers as integers one by one */
    while (fscanf(fp, "%d", &i) != EOF) {
        A[count++] = i;             // Add number to array
        fscanf(fp, "%s", buff);     // Read until whitespace
    }

    printf("\n%d\n", count);        // 10,000 ints in inp.txt

    for (int j = 0; j < count; j++) { printf("%d\n", A[j]); }    // Print A

    fclose(fp);
    
    /* Run Kernel */
    min<<<256, 256>>>(count, x);
    
    /* Wait for GPU to finish */
    hipDeviceSynchronize();
    
    /* Free Memory */
    hipFree(A);
}
