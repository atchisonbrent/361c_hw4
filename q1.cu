#include <stdio.h>
#include <stdlib.h>
#include <limits.h>
#include <hip/hip_runtime.h>

#define THREADS_PER_BLOCK 512
#define ARRAY_SIZE 512*512


__global__ void getMin(int* array, int* results, int n){
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if(i >= n){
        array[i] = INT_MAX;
    }
    __syncthreads();
    for(int s = blockDim.x/2; s > 0; s>>=1){
        if(i < ARRAY_SIZE){
            if(threadIdx.x < s){
                if(array[i] > array[i + s]){
                    array[i] = array[i + s];
                }
            }
        }
        __syncthreads();
    }
    if(threadIdx.x == 0){
        results[blockIdx.x] = array[i];
    }
}

__global__ void getMin2(int* array){
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    for(int s = blockDim.x/2; s > 0; s>>=1){
        if(i < s){
            if(threadIdx.x < s){
                if(array[i] > array[i + s]){
                    array[i] = array[i + s];
                }
            }
        }
        __syncthreads();
    }
}

/* Part B */
__global__ void last_digit(int n, int *A, int *B){
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < n; i += stride)
        B[i] = A[i] % 10;
}

int main(){
    FILE* fp;
    int temp;
    char buff[256];
    fp = fopen("inp.txt", "r");
    //int size = 256;
    int count = 0;
    int numBlocks = (ARRAY_SIZE/THREADS_PER_BLOCK);

    int* array = (int*)malloc(ARRAY_SIZE * sizeof(int));
    int* A = array;
    int* B = (int*)malloc(ARRAY_SIZE*sizeof(int));

    int* d_A;
    hipMalloc((void**)&d_A, ARRAY_SIZE*sizeof(int));
    int* d_B;
    hipMalloc((void**)&d_B, ARRAY_SIZE*sizeof(int));

    while(fscanf(fp, "%d", &temp) != EOF){
        array[count] = temp;
        count++;
        fscanf(fp, "%s", buff);
    }
    hipMemcpy(d_A, A, ARRAY_SIZE*sizeof(int), hipMemcpyHostToDevice);

    /* Kernel B */
    int blockSize = 256;
    int numBlocks2 = (count + blockSize - 1) / blockSize;
    last_digit<<<numBlocks2, blockSize>>>(count, d_A, d_B);

    int tempC = count;

    while(tempC < ARRAY_SIZE){
        array[tempC] = INT_MAX;
        tempC++;
    }

    int* d_array;
    hipMalloc((void **)&d_array, ARRAY_SIZE*sizeof(int));
    hipMemcpy(d_array, array, ARRAY_SIZE*sizeof(int), hipMemcpyHostToDevice);

    int* mid;
    hipMalloc((void **)&mid, numBlocks*sizeof(int));

    getMin<<<numBlocks, THREADS_PER_BLOCK>>>(d_array, mid, count);

    getMin2<<<16, 32>>>(mid);

    hipMemcpy(array, d_array, ARRAY_SIZE*sizeof(int), hipMemcpyDeviceToHost);

    int* h_mid = (int*)malloc(numBlocks*sizeof(int));
    hipMemcpy(h_mid, mid, numBlocks*sizeof(int), hipMemcpyDeviceToHost);

    for(int i = numBlocks - 1; i >= 0; i--){
        printf("%d\n", h_mid[i]);
    }

    // for(int i = ARRAY_SIZE - 1; i >= 0; i--){
    //     printf("%d\n", array[i]);
    // }
    
    /* Part A to File */
    FILE *f = fopen("q1a.txt", "w");
    fprintf(f, "%d", h_mid[0]);
    fclose(f);
    
    // int* newA = (int*)malloc(ARRAY_SIZE*sizeof(int));

    hipMemcpy(B, d_B, ARRAY_SIZE*sizeof(int), hipMemcpyDeviceToHost);

    /* Part B to File */
    FILE *f2 = fopen("q1b.txt", "w");
    for (int i = 0; i < count; i++) {
        fprintf(f2, "%d", B[i]);
        if (i + 1 != count) { fprintf(f2, ", "); }
    } fclose(f2);

    return 0;
}