
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <limits.h>

/* Part A */
__global__ void entries_in_range(int n, int *A, int *B){
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < n; i += stride) {
        if (0 <= A[i] && A[i] <= 99) { atomicAdd(B, 1); }
        if (100 <= A[i] && A[i] <= 199) { atomicAdd(&B[0], 1); }
        if (200 <= A[i] && A[i] <= 299) { atomicAdd(&B[1], 1); }
        if (300 <= A[i] && A[i] <= 399) { atomicAdd(&B[2], 1); }
        if (400 <= A[i] && A[i] <= 499) { atomicAdd(&B[3], 1); }
        if (500 <= A[i] && A[i] <= 599) { atomicAdd(&B[4], 1); }
        if (600 <= A[i] && A[i] <= 699) { atomicAdd(&B[5], 1); }
        if (700 <= A[i] && A[i] <= 799) { atomicAdd(&B[6], 1); }
        if (800 <= A[i] && A[i] <= 899) { atomicAdd(&B[7], 1); }
        if (900 <= A[i] && A[i] <= 999) { atomicAdd(&B[8], 1); }
    }
}

int main() {
    
    /* Open File */
    FILE *fp;
    fp = fopen("inp.txt", "r");
    
    char buff[256];
    const int M = 1<<20;
    const int d = 10;
    int *A = new int[M];
    int *B = new int[d];
    int *C = new int[d];
    int i, count = 0;
    
    /* Copy to GPU Memory */
    hipMallocManaged(&A, M * sizeof(int));
    hipMallocManaged(&B, d * sizeof(int));
    
    /* Read numbers as integers one by one */
    while (fscanf(fp, "%d", &i) != EOF) {
        A[count++] = i;             // Add number to array
        fscanf(fp, "%s", buff);     // Read until whitespace
    }
    
    /* Close FilePointer */
    fclose(fp);
    
    int blockSize = 256;
    int numBlocks = (count + blockSize - 1) / blockSize;
    entries_in_range<<<numBlocks, blockSize>>>(count, A, B);
    
    /* Wait for GPU */
    hipDeviceSynchronize();
    
    /* Print B */
    printf("Printing Array!\n");
    for (int i = 0; i < d; i++) {
        printf("%d", B[i]);
        if (i + 1 != d ) printf(", ");
    }
    printf("\n");
    
    /* Free Memory */
    hipFree(A);
    hipFree(B);
    
    return 0;
}
