
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <limits.h>

/* Part A */
__global__ void part_a(int n, int *A, int *B){
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < n; i += stride) {
        if (0 <= A[i] && A[i] <= 99) { atomicAdd(B, 1); }
        else if (100 <= A[i] && A[i] <= 199) { atomicAdd(&B[1], 1); }
        else if (200 <= A[i] && A[i] <= 299) { atomicAdd(&B[2], 1); }
        else if (300 <= A[i] && A[i] <= 399) { atomicAdd(&B[3], 1); }
        else if (400 <= A[i] && A[i] <= 499) { atomicAdd(&B[4], 1); }
        else if (500 <= A[i] && A[i] <= 599) { atomicAdd(&B[5], 1); }
        else if (600 <= A[i] && A[i] <= 699) { atomicAdd(&B[6], 1); }
        else if (700 <= A[i] && A[i] <= 799) { atomicAdd(&B[7], 1); }
        else if (800 <= A[i] && A[i] <= 899) { atomicAdd(&B[8], 1); }
        else if (900 <= A[i] && A[i] <= 999) { atomicAdd(&B[9], 1); }
    }
}

/* Part B */
__global__ void part_b(int n, int *A, int *B){
    __shared__ int s[10];
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < n; i += stride) {
        if (0 <= A[i] && A[i] <= 99) { atomicAdd(B, 1); }
        else if (100 <= A[i] && A[i] <= 199) { atomicAdd(&s[1], 1); }
        else if (200 <= A[i] && A[i] <= 299) { atomicAdd(&s[2], 1); }
        else if (300 <= A[i] && A[i] <= 399) { atomicAdd(&s[3], 1); }
        else if (400 <= A[i] && A[i] <= 499) { atomicAdd(&s[4], 1); }
        else if (500 <= A[i] && A[i] <= 599) { atomicAdd(&s[5], 1); }
        else if (600 <= A[i] && A[i] <= 699) { atomicAdd(&s[6], 1); }
        else if (700 <= A[i] && A[i] <= 799) { atomicAdd(&s[7], 1); }
        else if (800 <= A[i] && A[i] <= 899) { atomicAdd(&s[8], 1); }
        else if (900 <= A[i] && A[i] <= 999) { atomicAdd(&s[9], 1); }
    }
    __syncthreads();
    for (int i = 0; i < 10; i++) { atomicAdd(&B[i], s[i]); }
}

/* Part C */
__global__ void part_c(int *B, int *C){
    for (int i = 0; i < 10; i += 1) {
        int sum = 0;
        for (int j = 0; j < i; j++) { sum += B[j]; }
        C[i] += sum;
    }
}

int main() {
    
    /* Open File */
    FILE *fp;
    fp = fopen("inp.txt", "r");
    
    char buff[256];
    const int M = 1<<20;
    const int d = 10;
    int *A = new int[M];
    int *B = new int[d];
    int *B2 = new int[d];
    int *C = new int[d];
    int i, count = 0;
    
    /* Copy to GPU Memory */
    hipMallocManaged(&A, M * sizeof(int));
    hipMallocManaged(&B, d * sizeof(int));
    hipMallocManaged(&B2, d * sizeof(int));
    hipMallocManaged(&C, d * sizeof(int));
    
    /* Read numbers as integers one by one */
    while (fscanf(fp, "%d", &i) != EOF) {
        A[count++] = i;             // Add number to array
        fscanf(fp, "%s", buff);     // Read until whitespace
    }
    
    /* Close FilePointer */
    fclose(fp);
    
    /**************************************************/
    
    /* Part A */
    int blockSize = 256;
    int numBlocks = (count + blockSize - 1) / blockSize;
    part_a<<<numBlocks, blockSize>>>(count, A, B);
    
    /* Wait for GPU */
    hipDeviceSynchronize();
    
    /* Part A to File */
    FILE *f = fopen("q2a.txt", "w");
    for (int i = 0; i < d; i++) {
        fprintf(f, "%d", B[i]);
        if (i + 1 != d) { fprintf(f, ", "); }
    } fclose(f);
    
    /* Print B */
    printf("B: ");
    for (int i = 0; i < d; i++) {
        printf("%d", B[i]);
        if (i + 1 != d ) printf(", ");
    } printf("\n");
    
    /* Copy B to C */
    for (int i = 0; i < d; i++) { C[i] = B[i]; }
    
    /**************************************************/
    
    /* Part B */
    part_b<<<numBlocks, blockSize>>>(count, A, B2);
    
    /* Wait for GPU */
    hipDeviceSynchronize();
    
    /* Part B to File */
    FILE *f2 = fopen("q2b.txt", "w");
    for (int i = 0; i < d; i++) {
        fprintf(f2, "%d", B2[i]);
        if (i + 1 != d) { fprintf(f2, ", "); }
    } fclose(f2);
    
    /* Print B2 */
    printf("B2: ");
    for (int i = 0; i < d; i++) {
        printf("%d", B2[i]);
        if (i + 1 != d ) printf(", ");
    } printf("\n");
    
    /**************************************************/
    
    /* Part C */
    part_c<<<1, 1>>>(B, C);
    
    /* Wait for GPU */
    hipDeviceSynchronize();
    
    /* Part C to File */
    FILE *f3 = fopen("q2c.txt", "w");
    for (int i = 0; i < d; i++) {
        fprintf(f3, "%d", C[i]);
        if (i + 1 != d) { fprintf(f3, ", "); }
    } fclose(f3);
    
    /* Print C */
    printf("C: ");
    for (int i = 0; i < d; i++) {
        printf("%d", C[i]);
        if (i + 1 != d ) printf(", ");
    } printf("\n");
    
    /**************************************************/
    
    /* Free Memory */
    hipFree(A);
    hipFree(B);
    hipFree(B2);
    hipFree(C);
    
    return 0;
}
